#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/native/ConvUtils.h>

#include <algorithm>
#include <tuple>
#include <limits>

using namespace at;
using namespace native;

template <typename scalar_t, typename accscalar_t,
    int kKnownKernelT, int kKnownKernelH, int kKnownKernelW,
    int kKnownDilationT, int kKnownDilationH, int kKnownDilationW>
__global__ void conv_depthwise3d_cuda_kernel(
    const PackedTensorAccessor32<scalar_t, 5> input,
    PackedTensorAccessor32<scalar_t, 5> output,
    const PackedTensorAccessor32<scalar_t, 5> kernel,
    const scalar_t* bias,
    int strideT, int strideH, int strideW,
    int paddingT, int paddingH, int paddingW,
    int dilationT_, int dilationH_, int dilationW_)
{
  const int kT = kKnownKernelT > 0 ? kKnownKernelT : kernel.size(2);
  const int kH = kKnownKernelH > 0 ? kKnownKernelH : kernel.size(3);
  const int kW = kKnownKernelW > 0 ? kKnownKernelW : kernel.size(4);
  const int oC = output.size(1);
  const int oT = output.size(2);
  const int oH = output.size(3);
  const int oW = output.size(4);
  const int iC = input.size(1);
  const int iT = input.size(2);
  const int iH = input.size(3);
  const int iW = input.size(4);
  const int channel_multiplier = oC / iC;
  const int dilationT = kKnownDilationT > 0 ? kKnownDilationT : dilationT_;
  const int dilationH = kKnownDilationH > 0 ? kKnownDilationH : dilationH_;
  const int dilationW = kKnownDilationW > 0 ? kKnownDilationW : dilationW_;
  const int num_output = output.size(0) * output.stride(0);

  CUDA_KERNEL_LOOP(index, num_output) {
    const int out_col = index % oW;
    const int out_row = (index / oW) % oH;
    const int out_frame = (index / oW / oH) % oT;
    const int out_channel = (index / oW / oH / oT) % oC;
    const int batch = index / oW / oH / oT / oC;

    const int in_channel = out_channel / channel_multiplier;

    const int in_col_start = out_col * strideW - paddingW;
    const int in_row_start = out_row * strideH - paddingH;
    const int in_frame_start = out_frame * strideT - paddingT;

    accscalar_t sum = 0;
    const scalar_t *kernel_ptr = kernel[out_channel].data();
    const scalar_t *input_ptr =
        &input[batch][in_channel][in_frame_start][in_row_start][in_col_start];
    for (int k_frame = 0; k_frame < kT; ++k_frame) {
      const int in_frame = in_frame_start + k_frame * dilationT;
      for (int k_row = 0; k_row < kH; ++k_row) {
        const int in_row = in_row_start + k_row * dilationH;
        for (int k_col = 0; k_col < kW; ++k_col) {
          const accscalar_t op1 = *(kernel_ptr++);
          const int in_col = in_col_start + k_col * dilationW;
          if (in_frame >= 0 && in_row >= 0 && in_col >= 0 &&
              in_frame < iT && in_row < iH && in_col < iW) {
            sum += op1 * *(input_ptr);
          }
          input_ptr += dilationW;
        }
        input_ptr += iW * dilationH - kW * dilationW;
      }
      input_ptr += iW * (iH * dilationT - kH * dilationH);
    }
    if (bias != NULL) {
      sum += bias[out_channel];
    }

    output[batch][out_channel][out_frame][out_row][out_col] = sum;
  }
}

template <typename scalar_t, typename accscalar_t,
    int kKnownKernelT, int kKnownKernelH, int kKnownKernelW,
    int kKnownDilationT, int kKnownDilationH, int kKnownDilationW,
    int kKnownStrideT, int kKnownStrideH, int kKnownStrideW>
__global__ void
conv_depthwise3d_cuda_backward_input_kernel(
    const PackedTensorAccessor32<scalar_t, 5> grad_output,
    PackedTensorAccessor32<scalar_t, 5> grad_input,
    const PackedTensorAccessor32<scalar_t, 5> kernel,
    int strideT_, int strideH_, int strideW_,
    int paddingT, int paddingH, int paddingW,
    int dilationT_, int dilationH_, int dilationW_) {
  const int kT = kKnownKernelT > 0 ? kKnownKernelT : kernel.size(2);
  const int kH = kKnownKernelH > 0 ? kKnownKernelH : kernel.size(3);
  const int kW = kKnownKernelW > 0 ? kKnownKernelW : kernel.size(4);
  const int oC = grad_output.size(1);
  const int oT = grad_output.size(2);
  const int oH = grad_output.size(3);
  const int oW = grad_output.size(4);
  const int iC = grad_input.size(1);
  const int iT = grad_input.size(2);
  const int iH = grad_input.size(3);
  const int iW = grad_input.size(4);
  const int channel_multiplier = oC / iC;
  const int dilationT = kKnownDilationT > 0 ? kKnownDilationT : dilationT_;
  const int dilationH = kKnownDilationH > 0 ? kKnownDilationH : dilationH_;
  const int dilationW = kKnownDilationW > 0 ? kKnownDilationW : dilationW_;
  const int strideT = kKnownStrideT > 0 ? kKnownStrideT : strideT_;
  const int strideH = kKnownStrideH > 0 ? kKnownStrideH : strideH_;
  const int strideW = kKnownStrideW > 0 ? kKnownStrideW : strideW_;
  const int num_input = grad_input.size(0) * grad_input.stride(0);

  CUDA_KERNEL_LOOP(index, num_input) {
    const int in_col = index % iW;
    const int in_row = (index / iW) % iH;
    const int in_frame = (index / iW / iH) % iT;
    const int in_channel = (index / iW / iH / iT) % iC;
    const int batch = index / iW / iH / iT / iC;

    const int out_col_end = in_col + paddingW;
    const int out_row_end = in_row + paddingH;
    const int out_frame_end = in_frame + paddingT;

    const scalar_t* kernel_ptr = kernel[in_channel * channel_multiplier].data();
    accscalar_t sum = 0;

    for (int k_chn = in_channel * channel_multiplier;
         k_chn < (in_channel + 1) * channel_multiplier;
         ++k_chn) {
      const scalar_t* gout_ptr = grad_output[batch][k_chn].data();

      for (int k_frame = 0; k_frame < kT; ++k_frame) {
        const int out_frame_raw = out_frame_end - k_frame * dilationT;
        const int out_frame = out_frame_raw / strideT;
        for (int k_row = 0; k_row < kH; ++k_row) {
          const int out_row_raw = out_row_end - k_row * dilationH;
          const int out_row = out_row_raw / strideH;
          for (int k_col = 0; k_col < kW; ++k_col) {
            const accscalar_t op1 = *(kernel_ptr++);
            const int out_col_raw = out_col_end - k_col * dilationW;
            const int out_col = out_col_raw / strideW;

            const int out_offs = (out_frame * oH + out_row) * oW + out_col;

            accscalar_t op2 = (accscalar_t)0;
            if (out_col >= 0 && out_row >= 0 && out_frame >= 0 &&
                out_col < oW && out_row < oH && out_frame < oT) {
              op2 = *(gout_ptr + out_offs);
            }
            if (out_frame * strideT == out_frame_raw &&
                out_row * strideH == out_row_raw &&
                out_col * strideW == out_col_raw) {
              sum += op1 * op2;
            }
          }
        }
      }
    }

    grad_input[batch][in_channel][in_frame][in_row][in_col] = sum;
  }
}

template <typename scalar_t, typename accscalar_t,
    int kKnownStrideH, int kKnownStrideW>
__global__ void
conv_depthwise3d_cuda_backward_weight_kernel(
    const PackedTensorAccessor32<scalar_t, 5> grad_output,
    const PackedTensorAccessor32<scalar_t, 5> input,
    PackedTensorAccessor32<scalar_t, 5> grad_kernel,
    int strideT, int strideH_, int strideW_,
    int paddingT, int paddingH, int paddingW,
    int dilationT, int dilationH, int dilationW) {
  const int kC = grad_kernel.size(0);
  const int kT = grad_kernel.size(2);
  const int kH = grad_kernel.size(3);
  const int kW = grad_kernel.size(4);

  const int strideH = kKnownStrideH > 0 ? kKnownStrideH : strideH_;
  const int strideW = kKnownStrideW > 0 ? kKnownStrideW : strideW_;

  const int k_col = blockIdx.x % kW;
  const int k_row = (blockIdx.x / kW) % kH;
  const int k_frame = (blockIdx.x / kW / kH) % kT;
  const int k_channel = blockIdx.x / kW / kH / kT;
  scalar_t *result = &grad_kernel[k_channel][0][k_frame][k_row][k_col];

  const int oT = grad_output.size(2);
  const int oH = grad_output.size(3);
  const int oW = grad_output.size(4);
  const int iT = input.size(2);
  const int iH = input.size(3);
  const int iW = input.size(4);
  const int channel_multiplier = grad_output.size(1) / input.size(1);
  const int in_channel = k_channel / channel_multiplier;

  extern __shared__ int sdata_raw[];
  scalar_t* sdata = reinterpret_cast<scalar_t*>(sdata_raw);

  if (k_channel >= kC) {
    return;
  }

  const int laneid = threadIdx.x % C10_WARP_SIZE;
  const int warpid = threadIdx.x / C10_WARP_SIZE;
  const int nwarps = blockDim.x / C10_WARP_SIZE;

  accscalar_t grad = 0;
  int batch = warpid / oT;
  int gout_frame = warpid - batch * oT;
  for (int outer_pos = warpid; outer_pos < input.size(0) * oT;
       outer_pos += nwarps, gout_frame += nwarps) {
    while (gout_frame >= oT) { gout_frame -= oT; batch ++; }

    const int in_frame = (gout_frame * strideT) + (k_frame * dilationT) - paddingT;

    if (in_frame < 0 || in_frame >= iT) {
      continue;
    }

    const scalar_t* gout_ptr = grad_output[batch][k_channel][gout_frame].data() + laneid;
    const scalar_t* input_ptr = input[batch][in_channel][in_frame].data();

    int gout_row = laneid / oW;
    int gout_col = laneid - gout_row * oW;

    for (; gout_row < oH; ) {
      const accscalar_t op1 = *(gout_ptr);
      gout_ptr += C10_WARP_SIZE;

      const int in_col = (gout_col * strideW) + (k_col * dilationW) - paddingW;
      const int in_row = (gout_row * strideH) + (k_row * dilationH) - paddingH;
      const int in_pos = in_row * iW + in_col;

      accscalar_t op2 = (accscalar_t)0;
      if (in_col >= 0 && in_col < iW && in_row >= 0 && in_row < iH) {
        op2 = *(input_ptr + in_pos);
      }

      gout_col += C10_WARP_SIZE;
      while (gout_col >= oW) {
        gout_col -= oW; gout_row ++;
      }

      grad += op1 * op2;
    }
  }

  sdata[threadIdx.x] = grad;
  __syncthreads();

  CUDA_KERNEL_ASSERT(__popc(blockDim.x) == 1);
#pragma unroll
  for (int i = blockDim.x / 2; i >= 1; i >>= 1) {
    if (threadIdx.x < i) {
      sdata[threadIdx.x] += sdata[threadIdx.x + i];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    *result = sdata[0];
  }
}

template <int dim>
void conv_depthwise_shape_check(
    const Tensor& input,
    const Tensor& weight,
    const Tensor& bias,
    const Tensor& grad_output,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef dilation) {
  TORCH_CHECK(kernel_size.size() == dim,
              "kernel size length should be ", dim, ", but got ", kernel_size.size());
  TORCH_CHECK(stride.size() == dim,
              "stride length should be ", dim, ", but got ", stride.size());
  TORCH_CHECK(padding.size() == dim,
              "padding length should be ", dim, ", but got ", padding.size());
  TORCH_CHECK(dilation.size() == dim,
              "dilation length should be ", dim, ", but got ", dilation.size());

  TORCH_CHECK(weight.defined(),
              "Weight must be defined.");
  TORCH_CHECK(input.dim() == dim + 1 || input.dim() == dim + 2,
              "Input dimension should be ",
              dim + 1, "D or ", dim + 2, "D, got ",
              input.dim(), "D");
  TORCH_CHECK(weight.dim() == dim + 2,
              "Weight dimension should be ", dim + 2, "D, got ", weight.dim(), "D");
  TORCH_CHECK(weight.size(1) == 1,
              "Depthwise weight should have in_channels=1, got ", weight.size(1));
  TORCH_CHECK(weight.size(0) % input.size(-dim - 1) == 0,
              "Depthwise out channels should be a multiple of in channels, got ",
              weight.size(0), " and ", input.size(-dim - 1));
  for (int i = 0; i < dim; ++i) {
    TORCH_CHECK(weight.size(i + 2) == kernel_size[i],
                "kernel size and weight size mismatch, got ",
                kernel_size, " and ", weight.sizes());
    TORCH_CHECK(stride[i] >= 1,
                "stride should be at least 1, got ", stride);
    TORCH_CHECK(padding[i] >= 0,
                "padding should be non-negative, got ", padding);
    TORCH_CHECK(dilation[i] >= 1,
                "dilation should be at least 1, got ", dilation);
  }

  if (bias.defined()) {
    TORCH_CHECK(bias.dim() == 1,
                "Bias should be 1D tensor, got ", bias.dim(), "D");
    TORCH_CHECK(bias.size(0) == weight.size(0),
                "Bias length should be equal to out_channels, got ",
                bias.size(0), " and ", weight.size(0));
  }

  if (grad_output.defined()) {
    auto expected_output_size = conv_output_size(input.sizes(), weight.sizes(),
                                                 padding, stride, dilation);
    TORCH_CHECK(grad_output.dim() == expected_output_size.size(),
                "Expect grad_output to be ",
                expected_output_size.size(), "D, got ",
                grad_output.dim(), "D.");
    for (int i = 0; i < grad_output.dim(); ++i) {
      TORCH_CHECK(grad_output.size(i) == expected_output_size[i],
                  "Expect grad_output to be of same shape as output, got ",
                  grad_output.size(i), " and ", expected_output_size[i],
                  " at dimension ", i);
    }
  }
}


#define NODEF_OR_EQUAL(x, y) ((y) < 0 || (x) == (y))
#define NODEF_OR_EQUAL_3(x, y1, y2, y3) \
  (NODEF_OR_EQUAL(x[0], y1) && \
   NODEF_OR_EQUAL(x[1], y2) && \
   NODEF_OR_EQUAL(x[2], y3))

#define DWCONV3D_FORWARD_DISPATCH_SPECIALIZATION(kt, kh, kw, dilt, dilh, dilw) \
  if (NODEF_OR_EQUAL_3(kernel_size, (kt), (kh), (kw)) &&                    \
      NODEF_OR_EQUAL_3(dilation, (dilt), (dilh), (dilw))) {                 \
    using accscalar_t = acc_type<scalar_t, true>;                           \
    conv_depthwise3d_cuda_kernel                                            \
    <scalar_t, accscalar_t, (kt), (kh), (kw), (dilt), (dilh), (dilw)>       \
      <<<grid, block, (smem), at::cuda::getCurrentCUDAStream()>>>(          \
        input_.packed_accessor32<scalar_t, 5>(),                            \
        output_.packed_accessor32<scalar_t, 5>(),                           \
        weight_.packed_accessor32<scalar_t, 5>(),                           \
        bias_ptr,                                                           \
        stride[0], stride[1], stride[2],                                    \
        padding[0], padding[1], padding[2],                                 \
        dilation[0], dilation[1], dilation[2]);                             \
  } else

#define DWCONV3D_FORWARD_DISPATCH_OTHERS \
  {                                      \
    using accscalar_t = acc_type<scalar_t, true>;                           \
    conv_depthwise3d_cuda_kernel                                            \
    <scalar_t,accscalar_t, -1, -1, -1, -1, -1, -1>                          \
      <<<grid, block, (smem), at::cuda::getCurrentCUDAStream()>>>(          \
        input_.packed_accessor32<scalar_t, 5>(),                            \
        output_.packed_accessor32<scalar_t, 5>(),                           \
        weight_.packed_accessor32<scalar_t, 5>(),                           \
        bias_ptr,                                                           \
        stride[0], stride[1], stride[2],                                    \
        padding[0], padding[1], padding[2],                                 \
        dilation[0], dilation[1], dilation[2]);                             \
  }

Tensor conv_depthwise3d_cuda(
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    const Tensor& bias,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef dilation) {
  TORCH_CHECK(input.device() == weight.device(), "expects input and weight tensors to be on the same device.");
  if (bias.defined()) {
    TORCH_CHECK(input.device() == bias.device(), "expects input and bias tensors to be on the same device.");
  }

  conv_depthwise_shape_check<3>(input, weight, bias, Tensor() /* undefined */,
                                kernel_size, stride, padding, dilation);

  Tensor input_ = input.contiguous();

  if (input.dim() == 4 /* no batch */) {
    input_ = input.unsqueeze(0);
  }

  auto output_size = conv_output_size(input_.sizes(), weight.sizes(),
                                      padding, stride, dilation);
  for (size_t i = 0; i < output_size.size(); ++i) {
    TORCH_CHECK(output_size[i] > 0,
                "Output size should be positive, got ", output_size[i], " at dim ", i);
  }
  Tensor output = at::empty(output_size, input.options());
  Tensor output_ = output;
  Tensor weight_ = weight.contiguous();
  Tensor bias_ = bias.defined() ? bias.contiguous() : bias;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(),
      "conv_depthwise3d",
      [&]{
        int64_t num_outputs = output_.numel();
        int64_t block = 256;
        int64_t grid = std::min((num_outputs - 1) / block + 1, (int64_t)65536);
        int64_t smem = 0;

        const scalar_t* bias_ptr =
            bias_.defined() ? bias_.data_ptr<scalar_t>() : NULL;

        // Range check to avoid overflow in CUDA kernels.
        TORCH_CHECK(input_.numel() <= std::numeric_limits<int32_t>::max(),
                    "Input tensor is too large.");
        TORCH_CHECK(output_.numel() <= std::numeric_limits<int32_t>::max(),
                    "Output tensor is too large.");
        TORCH_CHECK(weight_.numel() <= std::numeric_limits<int32_t>::max(),
                    "Weight tensor is too large.");
        for (int i = 0; i < 3; ++i) {
          TORCH_CHECK(padding[i] * 2 + input.size(i + 2) <= std::numeric_limits<int32_t>::max(),
                      "Padded input tensor is too large.");
        }

        DWCONV3D_FORWARD_DISPATCH_SPECIALIZATION(3, 3, 3, 1, 1, 1)
        DWCONV3D_FORWARD_DISPATCH_SPECIALIZATION(-1, -1, -1, 1, 1, 1)
        DWCONV3D_FORWARD_DISPATCH_OTHERS
      }
  );

  return output;
}

#undef DWCONV3D_FORWARD_DISPATCH_SPECIALIZATION
#undef DWCONV3D_FORWARD_DISPATCH_OTHERS

#define DWCONV3D_BACKWARD_INPUT_DISPATCH_SPECIALIZATION(                    \
    kt, kh, kw, dilt, dilh, dilw, dt, dh, dw)                               \
  if (NODEF_OR_EQUAL_3(kernel_size, (kt), (kh), (kw)) &&                    \
      NODEF_OR_EQUAL_3(dilation, (dilt), (dilh), (dilw)) &&                 \
      NODEF_OR_EQUAL_3(stride, (dt), (dh), (dw))) {                         \
    using accscalar_t = acc_type<scalar_t, true>;                           \
    conv_depthwise3d_cuda_backward_input_kernel                             \
    <scalar_t, accscalar_t, (kt), (kh), (kw), (dilt), (dilh), (dilw), (dt), (dh), (dw)>  \
      <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(               \
        grad_output_.packed_accessor32<scalar_t, 5>(),                      \
        grad_input_.packed_accessor32<scalar_t, 5>(),                       \
        weight_.packed_accessor32<scalar_t, 5>(),                           \
        stride[0], stride[1], stride[2],                                    \
        padding[0], padding[1], padding[2],                                 \
        dilation[0], dilation[1], dilation[2]);                             \
  } else

#define DWCONV3D_BACKWARD_INPUT_DISPATCH_OTHERS                             \
  {                                                                         \
    using accscalar_t = acc_type<scalar_t, true>;                           \
    conv_depthwise3d_cuda_backward_input_kernel                             \
    <scalar_t, accscalar_t, -1, -1, -1, -1, -1, -1, -1, -1, -1>             \
      <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(               \
        grad_output_.packed_accessor32<scalar_t, 5>(),                      \
        grad_input_.packed_accessor32<scalar_t, 5>(),                       \
        weight_.packed_accessor32<scalar_t, 5>(),                           \
        stride[0], stride[1], stride[2],                                    \
        padding[0], padding[1], padding[2],                                 \
        dilation[0], dilation[1], dilation[2]);                             \
  }

#define DWCONV3D_BACKWARD_WEIGHT_DISPATCH_SPECIALIZATION(dh, dw)            \
  if (NODEF_OR_EQUAL_3(stride, -1, (dh), (dw))) {                           \
    using accscalar_t = acc_type<scalar_t, true>;                           \
    conv_depthwise3d_cuda_backward_weight_kernel                            \
    <scalar_t, accscalar_t, (dh), (dw)>                                     \
      <<<grid, block, smem, at::cuda::getCurrentCUDAStream()>>>(            \
        grad_output_.packed_accessor32<scalar_t, 5>(),                      \
        input_.packed_accessor32<scalar_t, 5>(),                            \
        grad_weight.packed_accessor32<scalar_t, 5>(),                       \
        stride[0], stride[1], stride[2],                                    \
        padding[0], padding[1], padding[2],                                 \
        dilation[0], dilation[1], dilation[2]);                             \
  } else

#define DWCONV3D_BACKWARD_WEIGHT_DISPATCH_OTHERS                            \
  {                                                                         \
    using accscalar_t = acc_type<scalar_t, true>;                           \
    conv_depthwise3d_cuda_backward_weight_kernel                            \
    <scalar_t, accscalar_t, -1, -1>                                         \
      <<<grid, block, smem, at::cuda::getCurrentCUDAStream()>>>(            \
        grad_output_.packed_accessor32<scalar_t, 5>(),                      \
        input_.packed_accessor32<scalar_t, 5>(),                            \
        grad_weight.packed_accessor32<scalar_t, 5>(),                       \
        stride[0], stride[1], stride[2],                                    \
        padding[0], padding[1], padding[2],                                 \
        dilation[0], dilation[1], dilation[2]);                             \
  }

std::tuple<Tensor&, Tensor&, Tensor&> _depthwise_3d_backward_cuda_out(
    Tensor& grad_input,
    Tensor& grad_weight,
    Tensor& grad_bias,
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef dilation,
    const std::array<bool, 3> output_mask)
{

  TORCH_CHECK(grad_output.device() == input.device() &&
              input.device() == weight.device(),
              "expects input, weight and grad_output to be on the same device.");
  conv_depthwise_shape_check<3>(
      input, weight, Tensor() /* undefined */, grad_output,
      kernel_size, stride, padding, dilation);

  const Tensor grad_output_ = grad_output.contiguous();
  const Tensor input_ = input.contiguous();
  const Tensor weight_ = weight.contiguous();

  Tensor grad_input_ =
      (output_mask[0] ?  grad_input
                      : Tensor());

  if (output_mask[0]) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_output.scalar_type(),
        "conv_depthwise3d",
        [&] {
          int64_t num_inputs = grad_input_.numel();
          int64_t block = 256;
          int64_t grid = std::min((num_inputs - 1) / block + 1, (int64_t)65536);

          // Range check to avoid overflow in CUDA kernels.
          TORCH_CHECK(grad_input_.numel() <= std::numeric_limits<int32_t>::max(),
                      "Input tensor is too large.");
          TORCH_CHECK(grad_output_.numel() <= std::numeric_limits<int32_t>::max(),
                      "Output tensor is too large.");
          TORCH_CHECK(weight_.numel() <= std::numeric_limits<int32_t>::max(),
                      "Weight tensor is too large.");
          for (int i = 0; i < 3; ++i) {
            TORCH_CHECK(padding[i] * 2 + input.size(i + 2) <= std::numeric_limits<int32_t>::max(),
                        "Padded input tensor is too large.");
          }

          DWCONV3D_BACKWARD_INPUT_DISPATCH_SPECIALIZATION(
              3, 3, 3, 1, 1, 1, 1, 1, 1)
          DWCONV3D_BACKWARD_INPUT_DISPATCH_SPECIALIZATION(
              3, 3, 3, 1, 1, 1, -1, -1, -1)
          DWCONV3D_BACKWARD_INPUT_DISPATCH_SPECIALIZATION(
              3, 3, 3, -1, -1, -1, 1, 1, 1)
          DWCONV3D_BACKWARD_INPUT_DISPATCH_SPECIALIZATION(
              3, 3, 3, -1, -1, -1, -1, -1, -1)
          DWCONV3D_BACKWARD_INPUT_DISPATCH_OTHERS
        }
    );
  }

  if (output_mask[1]) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_output.scalar_type(),
        "conv_depthwise3d",
        [&] {
          int64_t grid = grad_weight.numel();
          int64_t block = 256;
          int64_t smem = sizeof(scalar_t) * block;

          const int64_t int_max = std::numeric_limits<int32_t>::max();
          TORCH_CHECK(grad_input_.numel() <= int_max,
                      "Input tensor is too large.");
          TORCH_CHECK(grad_output_.numel() <= int_max,
                      "Output tensor is too large.");
          TORCH_CHECK(weight_.numel() <= int_max,
                      "Weight tensor is too large.");
          for (int i = 0; i < 3; ++i) {
            TORCH_CHECK(padding[i] * 2 + input.size(i + 2) <= int_max,
                        "Padded input tensor is too large.");
          }
          TORCH_CHECK(grad_output_.size(0) * grad_output_.size(2) < int_max - block / C10_WARP_SIZE &&
                      grad_output_.size(3) <= int_max - C10_WARP_SIZE &&
                      grad_output_.size(4) <= int_max - C10_WARP_SIZE,
                      "Output size is too large.");

          DWCONV3D_BACKWARD_WEIGHT_DISPATCH_SPECIALIZATION(1, 1)
          DWCONV3D_BACKWARD_WEIGHT_DISPATCH_SPECIALIZATION(2, 2)
          DWCONV3D_BACKWARD_WEIGHT_DISPATCH_OTHERS
        }
    );
  }

  if (output_mask[2]) {
    grad_bias = grad_output.sum({0, 2, 3, 4});
  }

  return std::tie(grad_input, grad_weight, grad_bias);

}


std::tuple<Tensor&, Tensor&, Tensor&> conv_depthwise3d_backward_cuda_out(
    Tensor& grad_input,
    Tensor& grad_weight,
    Tensor& grad_bias,
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef dilation) {
  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  return _depthwise_3d_backward_cuda_out(
      grad_input,
      grad_weight,
      grad_bias,
      grad_output,
      input,
      weight,
      kernel_size,
      stride,
      padding,
      dilation,
      {true,true,true});
}

std::tuple<Tensor, Tensor, Tensor> conv_depthwise3d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef dilation,
    const std::array<bool, 3> output_mask) {

  auto options = grad_output.options();
  Tensor grad_input =
      (output_mask[0] ? at::empty(input.sizes(), options) : Tensor());
  Tensor grad_weight =
      (output_mask[1] ? at::empty(weight.sizes(), options) : Tensor());
  Tensor grad_bias; /* undefined temporarily */

  return _depthwise_3d_backward_cuda_out(
      grad_input,
      grad_weight,
      grad_bias,
      grad_output,
      input,
      weight,
      kernel_size,
      stride,
      padding,
      dilation,
      output_mask
  );

}

#undef DWCONV3D_BACKWARD_INPUT_DISPATCH_SPECIALIZATION
#undef DWCONV3D_BACKWARD_INPUT_DISPATCH_OTHERS

#undef NODEF_OR_EQUAL_3
#undef NODEF_OR_EQUAL